#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include "../utils.h"

inline hipblasStatus_t
gemm(hipblasHandle_t handle,
     hipblasOperation_t transA, hipblasOperation_t transB,
     int m, int n, int k,
     const float* alpha,
     const half* A, int ldA,
     const half* B, int ldB,
     const float* beta,
     half* C, int ldC)
{
  return hipblasGemmEx(handle, transA, transB,
                      m, n, k,
                      reinterpret_cast<const float*>(alpha),
                      reinterpret_cast<const __half*>(A), HIP_R_16F, ldA,
                      reinterpret_cast<const __half*>(B), HIP_R_16F, ldB,
                      reinterpret_cast<const float*>(beta),
                      reinterpret_cast<      __half*>(C), HIP_R_16F, ldC,
                      HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}


void cublasMatmul(half* hA, half* hB, half* hC, int M, int N, int K){

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float alpha = 1.0;
    float beta = 0.0;

    half *dA;
    half *dB;
    half *dC;

    CUDA_CHECK(hipMalloc(&dA, M * K * 2));
    CUDA_CHECK(hipMalloc(&dB, K * N * 2));
    CUDA_CHECK(hipMalloc(&dC, M * N * 2));

    CUDA_CHECK(hipMemcpy(dA, hA, M * K * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dB, hB, K * N * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dC, hC, M * N * 2, hipMemcpyHostToDevice));

    // warmup
    for (int i = 0; i < 10; ++i)
    {
        gemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, K, &alpha, dA, K, dB, K, &beta, dC, M);
    }

    hipDeviceSynchronize();
    hipEventRecord(start);
    for (int i = 0; i < 200; ++i)
    {
        gemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, K, &alpha, dA, K, dB, K, &beta, dC, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "Running cost (ms) of CuBLAS is " << ms / 200.0 << "\n";
    std::cout << "TFLOPS: " << (float)M * N * K * 2 / (ms / 200.0) * 1e3 / 1e12 << "\n";

    hipMemcpy(hC, dC, M * K * 2, hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}