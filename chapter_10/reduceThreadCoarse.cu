
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#define N 4096 // N must be a power of 2
#define COARSE_FACTOR 4
#define BLOCK_SIZE 128

__global__
void reduceThreadCoarseKernel(float* A, float* sum){
    int i = threadIdx.x;
    int base = blockIdx.x * COARSE_FACTOR * 2 * BLOCK_SIZE;

    float curr = 0;
    for(int j=0;j<2*COARSE_FACTOR;++j){
        curr += A[base+j*BLOCK_SIZE+i];
    }

    __shared__ float sA[BLOCK_SIZE];
    sA[i] = curr;
    for(unsigned int stride = BLOCK_SIZE/2; stride >= 1; stride /= 2){
        __syncthreads();
        if(i < stride){
            sA[i] += sA[i+stride];
        }
    }
    if(i == 0){
        atomicAdd(sum, sA[0]);
    }
}

void reduceThreadCoarse(float* A, float* sum){
    float *A_d;
    hipMalloc((void**)&A_d, (N+1) * sizeof(float));
    hipMemcpy(A_d, A, N * sizeof(float), hipMemcpyHostToDevice);
    dim3 dimGrid(N/COARSE_FACTOR/2/BLOCK_SIZE, 1, 1);
    dim3 dimBlock(BLOCK_SIZE, 1, 1);
    reduceThreadCoarseKernel<<<dimGrid, dimBlock>>>(A_d, &A_d[N]);
    hipMemcpy(sum, &A_d[N], sizeof(float), hipMemcpyDeviceToHost);
    hipFree(A_d);
}

int main(){
    float *A, sum = 0;
    A = (float*)malloc( (N+1) * sizeof(float));

    srand(time(NULL));
    #pragma omp parallel for
    for(int i=0; i<N; i++){
        A[i] = (float)rand() / (float)(RAND_MAX);
    }
    A[N] = 0;
    reduceThreadCoarse(A, &sum);
    
    float curr = 0;
    #pragma omp parallel for
    for(int i=0; i<N; i++){
        curr += A[i];
    }
    if (abs(curr - sum) > 1e-2){
        std::cout<<"error: "<<curr<<' '<<sum<<std::endl;
    }
    
    free(A);

    return 0;
}