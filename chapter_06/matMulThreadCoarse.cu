
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>

#define N 100
#define M 200
#define K 300

#define TILE_SIZE 32
#define COARSE_SIZE ((K-1+TILE_SIZE)/TILE_SIZE)
__global__
void matMulThreadCoarseKernel(float* A, float* B, float* C){
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    // blockDim.x = blockDim.y = TILE_SIZE
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int taili = threadIdx.y;
    int tailj = threadIdx.x;

    float sums[COARSE_SIZE];
    for(int c=0;c<COARSE_SIZE;++c){
        sums[c] = 0;
    }
    for(int k=0;k<ceil(1.0*M/TILE_SIZE);++k){
        if(i<N && (k*TILE_SIZE+tailj)<M) As[taili][tailj] = A[i*M + k*TILE_SIZE + tailj];
        else As[taili][tailj] = 0;
        for(int c=0;c<COARSE_SIZE;++c){
            int j = tailj + c * TILE_SIZE;
            if((k*TILE_SIZE+taili)<M && j<K) Bs[taili][tailj] = B[(k*TILE_SIZE + taili)*K + j];
            else Bs[taili][tailj] = 0;
            __syncthreads();
            for(int p=0;p<TILE_SIZE;++p){
                sums[c] += As[taili][p] * Bs[p][tailj];
            }
            __syncthreads();
        }
    }
    for(int c=0;c<COARSE_SIZE;++c){
        int j = tailj + c * TILE_SIZE;
        if(i<N && j<K) C[i*K + j] = sums[c];
    }
}

void matMulThreadCoarse(float* A, float* B, float* C){
    float *A_d, *B_d, *C_d;
    hipMalloc((void**)&A_d, N * M * sizeof(float));
    hipMalloc((void**)&B_d, M * K * sizeof(float));
    hipMalloc((void**)&C_d, N * K * sizeof(float));
    hipMemcpy(A_d, A, N * M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, M * K * sizeof(float), hipMemcpyHostToDevice);
    dim3 dimGrid(1, ceil(1.0*N/TILE_SIZE), 1);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);
    matMulThreadCoarseKernel<<<dimGrid, dimBlock>>>(A_d, B_d, C_d);
    hipMemcpy(C, C_d, N * K * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main(){
    float *A, *B ,*C;
    A = (float*)malloc(N * M * sizeof(float));
    B = (float*)malloc(M * K * sizeof(float));
    C = (float*)malloc(N * K * sizeof(float));

    srand(time(NULL));
    #pragma omp parallel for
    for(int i=0; i<N*M; i++){
        A[i] = (float)rand() / (float)(RAND_MAX);
    }
    #pragma omp parallel for
    for(int i=0; i<M*K; i++){
        B[i] = (float)rand() / (float)(RAND_MAX);
    }
    
    matMulThreadCoarse(A, B, C);
    
    #pragma omp parallel for
    for(int i=0; i<N; i++){
        for(int j=0; j<K; j++){
            float sum = 0;
            for(int k=0;k<M;++k){
                sum += A[i*M+k] * B[k*K+j];
            }
            // in fact, no need to compare, see artical below, especially 5.4 
            // https://docs.nvidia.com/cuda/floating-point/#verifying-gpu-results
            if (abs(C[i*K+j] - sum) > 1e-4){
                std::cout<<i<<' '<<j<<' '<<C[i*K+j]<<' '<<sum<<' '<<abs(C[i*K+j] - sum)<<std::endl;
                break;
            }
        }

    }
    
    free(A);
    free(B);
    free(C);

    return 0;
}